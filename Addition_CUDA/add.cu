
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void sumKernel(int *result) {
    int index = threadIdx.x + 1; 
    atomicAdd(result, index);
    if (index==1){
        printf("Kernel => Sum = %d\n", *result);
    }
}
extern "C" {
int kernel_launcher(int i){
    int *result;
    int *d_result;
    int sum = 0;

    result = &sum;

    hipMalloc((void **)&d_result, sizeof(int));
    hipMemcpy(d_result, result, sizeof(int), hipMemcpyHostToDevice);
    sumKernel<<<1, i>>>(d_result);
    hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_result);
    printf("C => Sum = %d\n", *result);

    return *result;
}
}