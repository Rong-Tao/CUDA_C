#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

// CUDA Kernel for vector addition
__global__ void sumKernel(float *a, float *b, float *c, int height, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < height && col < width) {
        int index = row * width + col;
        c[index] = a[index] + b[index];
    }
}


// Host function to launch the kernel
extern "C" void kernel_launcher(float *a, float *b, float *c, int height, int width) {
    dim3 blockSize(16, 16); // 16x16 threads per block
    dim3 numBlocks((width + blockSize.x - 1) / blockSize.x, 
                   (height + blockSize.y - 1) / blockSize.y);

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, width * height * sizeof(float));
    hipMalloc(&d_b, width * height * sizeof(float));
    hipMalloc(&d_c, width * height * sizeof(float));

    hipMemcpy(d_a, a, width * height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, width * height * sizeof(float), hipMemcpyHostToDevice);

    sumKernel<<<numBlocks, blockSize>>>(d_a, d_b, d_c, height, width);

    hipMemcpy(c, d_c, width * height * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
