#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;


__global__ void findIntersection(int *list1, int *list2, int list1Size, int list2Size, int deltax, int deltay, int *result) {
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx_x < list1Size && idx_y < list2Size) {
        int x = list1[idx_x * 2] + deltax;
        int y = list1[idx_x * 2 + 1] + deltay;

        if (x == list2[idx_y * 2] && y == list2[idx_y * 2 + 1]) {
            result[idx_x] += 1; // Mark as found
        }
    }
}



extern "C" void launchFindIntersection(int *h_list1, int *h_list2, int list1Size, int list2Size, int deltax, int deltay, int *h_result) {
    int *d_list1, *d_list2, *d_result;
    int size1 = list1Size * 2 * sizeof(int);
    int size2 = list2Size * 2 * sizeof(int);
    int resultSize = list1Size * sizeof(int);

    hipMalloc((void**)&d_list1, size1);
    hipMalloc((void**)&d_list2, size2);
    hipMalloc((void**)&d_result, resultSize);

    hipMemcpy(d_list1, h_list1, size1, hipMemcpyHostToDevice);
    hipMemcpy(d_list2, h_list2, size2, hipMemcpyHostToDevice);
    hipMemset(d_result, 0, resultSize); 


    dim3 blockSize(16, 16); // Example block size, adjust as needed
    dim3 numBlocks((list1Size + blockSize.x - 1) / blockSize.x,
                   (list2Size + blockSize.y - 1) / blockSize.y);

    // Launch the kernel
    findIntersection<<<numBlocks, blockSize>>>(d_list1, d_list2, list1Size, list2Size, deltax, deltay, d_result);
    hipDeviceSynchronize();


    hipMemcpy(h_result, d_result, resultSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_list1);
    hipFree(d_list2);
    hipFree(d_result);
}